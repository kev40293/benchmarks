#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

__global__ void add(int* a, int * b, int *c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void add_double(double *a, double *b, double*c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void bench_gpu_memory() {
	clock_t start, stop;
	int *d_c, *d_cd; // device copies
	// Memory benchmarks
	for (int i=2; i < 3; i++) {
		int bsize = pow(1024, i);
		int wsize = 1024*1024*512;
		double rate;
		char * moo = (char*)malloc(bsize);
		hipMalloc((void**) &d_c, 1024*1024);
		hipMalloc((void**) &d_cd, 1024*1024);
		printf("%d\n", wsize);
/*
		start = clock();
		for (int k =0; k < wsize; k++) {
			hipMemcpy(d_c, (char*) moo, bsize, hipMemcpyHostToDevice);
		}
		stop = clock();
		rate = (wsize * bsize / (1024*1024)) / ((stop-start) / 1000000.0);
		printf("Host to device, bs =  %d, bandwidth = %Lf MB/s\n", bsize, rate);

		start = clock();
		for (int k =0; k < wsize; k++) {
			hipMemcpy(d_c, (char*) moo, bsize, hipMemcpyDeviceToHost);
		}
		stop = clock();
		rate = (wsize * bsize / (1024*1024)) / ((stop-start) / 1000000.0);
		printf("device to host, bs =  %d, bandwidth = %lf MB/s\n", bsize, rate);
*/

		start = clock();
		for (int k =0; k < wsize; k++) {
			hipMemcpy(d_c, d_cd, bsize, hipMemcpyDeviceToDevice);
		}
		stop = clock();
		printf("%d\n", stop-start);
		rate = (wsize * bsize / (1024)) / ((stop-start) / 1000000.0);
		printf("device to device, bs =  %d, bandwidth = %lf KB/s\n", bsize, rate);
		hipFree(d_c);
		hipFree(d_cd);
		free(moo);
	}

}

void bench_gpu_ops() {
	int prob_size = 1000;
	int a[prob_size],b[prob_size]; // Host copies
	double ad[prob_size], bd[prob_size];
	clock_t start, stop;
	int *d_a, *d_b, *d_c; // device copies
	double *d_ad, *d_bd, *d_cd;

	int size = sizeof(int) * prob_size;
	int sized = sizeof(double) * prob_size;

	// Allocate space
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);
	for (int i = 0; i < prob_size; i++) {
		a[i] = i;
		ad[i] = i;
		b[i] = prob_size - i;
		bd[i] = prob_size - i;
	}
	// Copy inputs to dev
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	start = clock();
	for (int i = 0; i < 1000000; i++) {
		add<<<1,1>>>(d_a, d_b, d_c);
	}
	stop = clock();
	int iop_time = stop-start;
	hipFree(d_a);
	hipFree(d_b);

	hipMalloc((void **) &d_ad, sized);
	hipMalloc((void **) &d_bd, sized);
	hipMalloc((void **) &d_cd, sized);
	printf("Time taken (integer) = %d ms\n", iop_time);
	printf("GPU IOPs = %f\n", 1000000.0*prob_size / (iop_time/1000000.0));

	hipMemcpy(d_ad, &ad, sized, hipMemcpyHostToDevice);
	hipMemcpy(d_bd, &bd, sized, hipMemcpyHostToDevice);
	start = clock();
	for (int i = 0; i < 1000000; i++) {
		add_double<<<1,1>>>(d_ad, d_bd, d_cd);
	}
	stop = clock();
	int flop_time = stop-start;
	//printf("Our result is %d\n", c[0]);
	hipFree(d_ad);
	hipFree(d_bd);
	hipFree(d_cd);
	printf("Time taken (double) = %d ms\n", flop_time);
	printf("GPU FLOPs = %f\n", 1000000.0*prob_size / (flop_time/1000000.0));
}

int main(int argc, char** argv) {
	if (argc > 1) {
		switch (argv[1][0]) {
			case 'g':
				bench_gpu_ops(); break;
			case 'm':
				bench_gpu_memory(); break;
		}
	}

	return 0;
}
